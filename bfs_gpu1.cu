#include <iostream>
#include <vector>
#include <queue>
#include <climits>
#include <fstream>
#include <sstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

// CUDA kernel for parallel neighbor processing
__global__ void process_level_kernel(
    int* d_adjacency_list,
    int* d_adjacency_offsets,
    int* d_distances,
    int* d_frontier,
    int* d_new_frontier,
    int* d_frontier_size,
    int* d_new_frontier_size,
    int current_depth
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= *d_frontier_size) return;

    int current = d_frontier[tid];
    int start = d_adjacency_offsets[current];
    int end = d_adjacency_offsets[current + 1];

    for (int i = start; i < end; i++) {
        int neighbor = d_adjacency_list[i];
        if (d_distances[neighbor] == INT_MAX) {
            d_distances[neighbor] = current_depth + 1;
            int idx = atomicAdd(d_new_frontier_size, 1);
            d_new_frontier[idx] = neighbor;
        }
    }
}

void BFS_GPU(const std::vector<std::vector<int>>& graph, int source, int branching_factor) {
    auto start_time = std::chrono::high_resolution_clock::now();
    
    int n = graph.size();
    
    // Convert graph to CSR format
    std::vector<int> adjacency_list;
    std::vector<int> adjacency_offsets(n + 1, 0);
    
    for (int i = 0; i < n; i++) {
        adjacency_offsets[i + 1] = adjacency_offsets[i] + graph[i].size();
        for (int neighbor : graph[i]) {
            adjacency_list.push_back(neighbor);
        }
    }

    // Allocate device memory
    int *d_adjacency_list, *d_adjacency_offsets, *d_distances;
    int *d_frontier, *d_new_frontier;
    int *d_frontier_size, *d_new_frontier_size;
    
    hipMalloc(&d_adjacency_list, adjacency_list.size() * sizeof(int));
    hipMalloc(&d_adjacency_offsets, (n + 1) * sizeof(int));
    hipMalloc(&d_distances, n * sizeof(int));
    hipMalloc(&d_frontier, n * sizeof(int));
    hipMalloc(&d_new_frontier, n * sizeof(int));
    hipMalloc(&d_frontier_size, sizeof(int));
    hipMalloc(&d_new_frontier_size, sizeof(int));

    // Initialize host arrays
    std::vector<int> distances(n, INT_MAX);
    distances[source] = 0;
    std::vector<int> frontier = {source};
    int frontier_size = 1;
    int new_frontier_size = 0;
    
    // Copy data to device
    hipMemcpy(d_adjacency_list, adjacency_list.data(), adjacency_list.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjacency_offsets, adjacency_offsets.data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_distances, distances.data(), n * sizeof(int), hipMemcpyHostToDevice);
    
    int current_depth = 0;
    int max_depth = 0;
    int nodes_visited = 1;

    // BFS iterations
    while (frontier_size > 0) {
        hipMemcpy(d_frontier, frontier.data(), frontier_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_frontier_size, &frontier_size, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_new_frontier_size, &new_frontier_size, sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel
        int block_size = 256;
        int num_blocks = (frontier_size + block_size - 1) / block_size;
        process_level_kernel<<<num_blocks, block_size>>>(
            d_adjacency_list,
            d_adjacency_offsets,
            d_distances,
            d_frontier,
            d_new_frontier,
            d_frontier_size,
            d_new_frontier_size,
            current_depth
        );

        // Get new frontier size
        hipMemcpy(&new_frontier_size, d_new_frontier_size, sizeof(int), hipMemcpyDeviceToHost);
        
        // Get new frontier
        frontier.resize(new_frontier_size);
        hipMemcpy(frontier.data(), d_new_frontier, new_frontier_size * sizeof(int), hipMemcpyDeviceToHost);
        
        frontier_size = new_frontier_size;
        new_frontier_size = 0;
        
        current_depth++;
        if (frontier_size > 0) {
            max_depth = current_depth;
            nodes_visited += frontier_size;
        }
    }

    // Clean up
    hipFree(d_adjacency_list);
    hipFree(d_adjacency_offsets);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_new_frontier);
    hipFree(d_frontier_size);
    hipFree(d_new_frontier_size);

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    
    printf("%d,%d,GPU,%d,%.3f,%d,%d\n", 
           graph.size(),          // graph_size
           branching_factor,      // branching_factor
           source,               // source_node
           duration.count() / 1000.0,  // time_ms
           max_depth,            // max_depth
           nodes_visited);       // nodes_visited
}

std::vector<std::vector<int>> read_graph(std::ifstream& file) {
    std::string line;
    std::getline(file, line);
    int n = std::stoi(line);
    
    std::vector<std::vector<int>> graph(n);
    
    for (int i = 0; i < n; i++) {
        std::getline(file, line);
        std::istringstream iss(line);
        std::string vertex;
        iss >> vertex;
        
        int neighbor;
        while (iss >> neighbor) {
            graph[i].push_back(neighbor);
        }
    }
    
    std::getline(file, line);
    return graph;
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <branching_factor>\n";
        return 1;
    }
    
    const int branching_factor = std::stoi(argv[1]);
    
    auto total_start_time = std::chrono::high_resolution_clock::now();
    
    std::ifstream file("random_graphs.txt");
    if (!file.is_open()) {
        std::cerr << "Error: Could not open random_graphs.txt\n";
        return 1;
    }

    int graph_number = 1;
    int total_searches = 0;
    
    while (!file.eof()) {
        std::string peek;
        if (!std::getline(file, peek)) break;
        file.seekg(-peek.length()-1, std::ios::cur);
        
        std::vector<std::vector<int>> graph = read_graph(file);
        if (graph.empty()) break;
        
        std::cout << "\nGraph " << graph_number << " (Size: " << graph.size() << "):\n";
        
        BFS_GPU(graph, 0, branching_factor);
        BFS_GPU(graph, graph.size() / 2, branching_factor);
        
        graph_number++;
        total_searches += 2;
    }

    file.close();
    
    auto total_end_time = std::chrono::high_resolution_clock::now();
    auto total_duration = std::chrono::duration_cast<std::chrono::microseconds>(total_end_time - total_start_time);
    
    std::cout << "\nTotal Statistics:\n";
    std::cout << "Total time: " << total_duration.count() / 1000.0 << " milliseconds\n";
    std::cout << "Graphs processed: " << graph_number - 1 << "\n";
    std::cout << "Total searches performed: " << total_searches << "\n";
    std::cout << "Average time per search: " << (total_duration.count() / total_searches) / 1000.0 << " milliseconds\n";

    return 0;
}
