
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <ctime>
#include <fstream>

// Use specific using declarations for frequently used components
using std::vector;
using std::cout;
using std::cerr;
using std::endl;
using std::string;
using std::ofstream;
using std::ios;

// Generate a random graph with n vertices and approximately edge_density percentage of possible edges
vector<vector<int>> generate_random_graph(int n, double edge_density) {
    vector<vector<int>> graph(n);
    
    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> prob(0.0, 1.0);
    std::uniform_int_distribution<> vertex_dist(0, n-1);
    
    // For each pair of vertices
    for (int i = 0; i < n; i++) {
        for (int j = i + 1; j < n; j++) {
            // Add edge with probability edge_density
            if (prob(gen) < edge_density) {
                graph[i].push_back(j);
                graph[j].push_back(i); // Since it's an undirected graph
            }
        }
    }
    
    return graph;
}

// Save graph to file
void save_graph_to_file(const vector<vector<int>>& graph, const string& filename) {
    ofstream out(filename, ios::app);
    out << graph.size() << "\n"; // First line contains number of vertices
    
    // Write each adjacency list
    for (size_t i = 0; i < graph.size(); i++) {
        out << i << ": ";
        for (int neighbor : graph[i]) {
            out << neighbor << " ";
        }
        out << "\n";
    }
    out << "\n"; // Empty line between graphs
    out.close();
}

int main() {
    // Seed for reproducibility
    srand(time(0));
    
    // Clear the output file
    ofstream out("random_graphs.txt", ios::trunc);
    out.close();
    
    // Generate 10 larger graphs
    for (int i = 0; i < 10; i++) {
        // Random size between 100 and 1000 vertices
        int size = rand() % 901 + 100;  // 901 = (1000-100+1)
        
        // Random edge density between 0.01 and 0.1 (sparser for larger graphs)
        double density = (rand() % 9 + 1) / 100.0;
        
        // Generate and save the graph
        vector<vector<int>> graph = generate_random_graph(size, density);
        
        cout << "Generated graph " << i + 1 << " with " << size << " vertices"
             << " and density " << density << "\n";
             
        save_graph_to_file(graph, "random_graphs.txt");
    }
    
    cout << "\nAll graphs have been saved to 'random_graphs.txt'\n";
    return 0;
} 